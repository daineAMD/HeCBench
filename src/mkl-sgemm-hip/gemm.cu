#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <iterator>
#include <limits>
#include <list>
#include <vector>
#include <type_traits>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas/hipblas.h>
#include <x86intrin.h>

inline float half_to_float(hipblasHalf val)
{
#ifdef HIPBLAS_USE_HIP_HALF
    return __half2float(val);
#else
    return _cvtsh_ss(val);
#endif
}

inline hipblasHalf float_to_half(float val)
{
#ifdef HIPBLAS_USE_HIP_HALF
    return __float2half(val);
#else
    uint16_t a = _cvtss_sh(val, 0);
    return a;
#endif
}

template <typename T>
void print_2x2_matrix_values(T M, int ldM, std::string M_name)
{
  std::cout << std::endl;
  std::cout << "\t\t\t" << M_name << " = [ " << (float)M[0*ldM + 0] << ", " << (float)M[1*ldM + 0]         << ", ...\n";
  std::cout << "\t\t\t    [ "                << (float)M[0*ldM + 1] << ", " << (float)M[1*ldM + 1] << ", ...\n";
  std::cout << "\t\t\t    [ "                << "...\n";
  std::cout << std::endl;
}

template <>
void print_2x2_matrix_values(hipblasHalf* M, int ldM, std::string M_name)
{
    std::cout << std::endl;
    std::cout << "\t\t\t" << M_name << " = [ " << half_to_float(M[0*ldM + 0]) << ", " << half_to_float(M[1*ldM + 0])         << ", ...\n";
    std::cout << "\t\t\t    [ "                << half_to_float(M[0*ldM + 1]) << ", " << half_to_float(M[1*ldM + 1]) << ", ...\n";
    std::cout << "\t\t\t    [ "                << "...\n";
    std::cout << std::endl;
}

//
// helpers for initializing templated scalar data type values.
//
template <typename fp> void rand_matrix(fp *M, int n_row, int n_col)
{
  for (int i = 0; i < n_row; i++)
    for (int j = 0; j < n_col; j++)
      M[i * n_col + j] = rand() % 2;
}

template <>
void rand_matrix(hipblasHalf* M, int n_row, int n_col)
{
    for(int i = 0; i < n_row; i++)
        for(int j = 0; j < n_col; j++)
            M[i * n_col + j] = float_to_half(rand() % 2);
}

//
// Main example for Gemm consisting of
// initialization of A, B and C matrices as well as
// scalars alpha and beta.  Then the product
//
// C = alpha * op(A) * op(B) + beta * C
//
// is performed and finally the results are post processed.
//
template <typename fp>
void run_gemm_example(int m, int k, int n, int repeat) {

  //
  // Initialize data for Gemm
  //
  // C = alpha * op(A) * op(B)  + beta * C
  //

  // set scalar fp values
  const fp alpha = std::is_same_v<fp, hipblasHalf> ? fp(float_to_half(2.0)) : fp(2.0);
  const fp beta  = std::is_same_v<fp, hipblasHalf> ? fp(float_to_half(0.5)) : fp(0.5);

  const size_t A_size = sizeof(fp) * m * k;
  const size_t B_size = sizeof(fp) * k * n;
  const size_t C_size = sizeof(fp) * m * n;

  // prepare matrix data
  fp* a = (fp *) aligned_alloc(64, A_size);
  fp* b = (fp *) aligned_alloc(64, B_size);
  fp* c = (fp *) aligned_alloc(64, C_size);

  srand(2);
  rand_matrix(a, m, k);
  rand_matrix(b, k, n);
  rand_matrix(c, m, n);

  fp *da, *db, *dc;
  hipMalloc((void**)&da, A_size);
  hipMalloc((void**)&db, B_size);
  hipMalloc((void**)&dc, C_size);
  hipMemcpy(da, a, A_size, hipMemcpyHostToDevice);
  hipMemcpy(db, b, B_size, hipMemcpyHostToDevice);

  // create execution queue and buffers of matrix data
  hipblasHandle_t h;
  hipblasCreate(&h);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    if constexpr (std::is_same_v<fp, hipblasHalf>)
      hipblasHgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                   &alpha, db, n, da, k, &beta, dc, n);
    else if constexpr (std::is_same_v<fp, float>)
      hipblasSgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                   &alpha, db, n, da, k, &beta, dc, n);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average GEMM execution time: %f (us)\n", (time * 1e-3f) / repeat);

  hipMemcpy(c, dc, C_size, hipMemcpyDeviceToHost);
  hipblasDestroy(h);

  hipFree(da);
  hipFree(db);
  hipFree(dc);

  //
  // Post Processing
  //

  std::cout << "\n\t\tOutputting 2x2 block of A,B,C matrices:" << std::endl;

  // output the top 2x2 block of A matrix
  print_2x2_matrix_values(a, k, "A");

  // output the top 2x2 block of B matrix
  print_2x2_matrix_values(b, n, "B");

  // output the top 2x2 block of C matrix
  print_2x2_matrix_values(c, n, "C");

  free(a);
  free(b);
  free(c);
}

//
// Main entry point for example.
//
int main (int argc, char ** argv) {
  if (argc != 5) {
    printf("Usage: %s <m> <k> <n> <repeat>\n", argv[0]);
    return 1;
  }
  const int m = atoi(argv[1]);
  const int k = atoi(argv[2]);
  const int n = atoi(argv[3]);
  const int repeat = atoi(argv[4]);

  std::cout << "\tRunning with half precision data type:" << std::endl;
  run_gemm_example<hipblasHalf>(m, k, n, repeat);

  std::cout << "\tRunning with single precision data type:" << std::endl;
  run_gemm_example<float>(m, k, n, repeat);

  return 0;
}

